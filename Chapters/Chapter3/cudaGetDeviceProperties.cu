#include "hip/hip_runtime.h"
// File: hipGetDeviceProperties.cu
//
// Compiler Command:
// $ nvcc hipGetDeviceProperties.cu -o hipGetDeviceProperties

// Head files
#include <stdio.h>

#include <hip/hip_runtime.h>


// main function
int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
        (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev, driverVersion = 0, runtimeVersion = 0;
    dev =0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d: \"%s\"\n", dev, deviceProp.name);
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n",driverVersion/1000, (driverVersion%100)/10,runtimeVersion/1000, (runtimeVersion%100)/10);
    printf(" CUDA Capability Major/Minor version number: %d.%d\n",deviceProp.major, deviceProp.minor);
    printf(" Total amount of global memory: %.2f MBytes (%llu bytes)\n",(float)deviceProp.totalGlobalMem/(pow(1024.0,3)),(unsigned long long) deviceProp.totalGlobalMem);
    printf(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n",deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
    printf(" Memory Clock rate: %.0f Mhz\n",deviceProp.memoryClockRate * 1e-3f);
    printf(" Memory Bus Width: %d-bit\n",deviceProp.memoryBusWidth);
    if (deviceProp.l2CacheSize) {
        printf(" L2 Cache Size: %d bytes\n",
        deviceProp.l2CacheSize);
    }

    printf(" Max Texture Dimension Size (x,y,z) 1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
    deviceProp.maxTexture1D , deviceProp.maxTexture2D[0],
    deviceProp.maxTexture2D[1],
    deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
    deviceProp.maxTexture3D[2]);

    printf(" Max Layered Texture Size (dim) x layers 1D=(%d) x %d, 2D=(%d,%d) x %d\n",
    deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
    deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
    deviceProp.maxTexture2DLayered[2]);

    printf(" Total amount of constant memory: %lu bytes\n",deviceProp.totalConstMem);
    printf(" Total amount of shared memory per block: %lu bytes\n",deviceProp.sharedMemPerBlock);
    printf(" Total number of registers available per block: %d\n",deviceProp.regsPerBlock);
    printf(" Warp size: %d\n", deviceProp.warpSize);
    printf(" Maximum number of threads per multiprocessor: %d\n",deviceProp.maxThreadsPerMultiProcessor);
    printf(" Maximum number of threads per block: %d\n",deviceProp.maxThreadsPerBlock);

    printf(" Maximum sizes of each dimension of a block: %d x %d x %d\n",
    deviceProp.maxThreadsDim[0],
    deviceProp.maxThreadsDim[1],
    deviceProp.maxThreadsDim[2]);

    printf(" Maximum sizes of each dimension of a grid: %d x %d x %d\n",
    deviceProp.maxGridSize[0],
    deviceProp.maxGridSize[1],
    deviceProp.maxGridSize[2]);

    printf(" Maximum memory pitch: %lu bytes\n", deviceProp.memPitch);

    exit(EXIT_SUCCESS);
}


// For a system with multiple GPUs, it's necessary to choose one of these GPUs as our device.
// The stratage is that: the GPU with best performance must has the largest number of SMs.
// Implementation code listed below: 

// int numDevices = 0;
// hipGetDeviceCount(&numDevices);
// if (numDevices > 1) {
//     int maxMultiprocessors = 0, maxDevice = 0;
//     for (int device=0; device<numDevices; device++) {
//         hipDeviceProp_t props;
//         hipGetDeviceProperties(&props, device);
//         if (maxMultiprocessors < props.multiProcessorCount) {
//             maxMultiprocessors = props.multiProcessorCount;
//             maxDevice = device;
//         }
//     }
//     hipSetDevice(maxDevice);
// }


// Runnning result
// yangyang@yangyang-XPS-8900:~/Desktop/cudaCodeDebug/CodeDebug2$ ./hipGetDeviceProperties 
// ./hipGetDeviceProperties Starting...
// Detected 1 CUDA Capable device(s)
// Device 0: "GeForce GTX 750 Ti"
//  CUDA Driver Version / Runtime Version 9.1 / 9.1
//  CUDA Capability Major/Minor version number: 5.0
//  Total amount of global memory: 1.95 MBytes (2090598400 bytes)
//  GPU Clock rate: 1084 MHz (1.08 GHz)
//  Memory Clock rate: 2700 Mhz
//  Memory Bus Width: 128-bit
//  L2 Cache Size: 2097152 bytes
//  Max Texture Dimension Size (x,y,z) 1D=(65536), 2D=(65536,65536), 3D=(4096,4096,4096)
//  Max Layered Texture Size (dim) x layers 1D=(16384) x 2048, 2D=(16384,16384) x 2048
//  Total amount of constant memory: 65536 bytes
//  Total amount of shared memory per block: 49152 bytes
//  Total number of registers available per block: 65536
//  Warp size: 32
//  Maximum number of threads per multiprocessor: 2048
//  Maximum number of threads per block: 1024
//  Maximum sizes of each dimension of a block: 1024 x 1024 x 64
//  Maximum sizes of each dimension of a grid: 2147483647 x 65535 x 65535
//  Maximum memory pitch: 2147483647 bytes



